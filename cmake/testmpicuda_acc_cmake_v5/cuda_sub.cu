
#include <hip/hip_runtime.h>
#ifdef ENABLE_CUDA

#include <stdio.h>
#include <cuda_runtime.h>
#include "cuda_sub.h"
#include <iostream>
using namespace std;

void GetCudaDeviceCount( int &num_gpus )
{
    cudaGetDeviceCount( &num_gpus );

    for ( int i = 0; i < num_gpus; ++ i )
    {
        cudaDeviceProp dprop;
        cudaGetDeviceProperties(&dprop, i);
        printf("%d: %s\n", i, dprop.name);
    }
}

#endif